#include "hip/hip_runtime.h"
#include "../include/Net_preprocess.h"
#include <opencv2/opencv.hpp>

__global__ void Net_preprocess_kernel(uint8_t* left,uint8_t* right,int width, int height, float* dst)
{
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if ( position>= width*height) 
        return;

    int dx = position % width;
    int dy = position / width;

    float left_c0, left_c1, left_c2;
    float right_c0, right_c1, right_c2;

    uint8_t* v_left = left + (dy * width + dx) * 3;
    uint8_t* v_right= right + (dy * width + dx) * 3;

    left_c0 = (float)v_left[0]/255.0;
    left_c1 = (float)v_left[1]/255.0;
    left_c2 = (float)v_left[2]/255.0;

    right_c0 = (float)v_right[0]/255.0;
    right_c1 = (float)v_right[1]/255.0;
    right_c2 = (float)v_right[2]/255.0;


    //bgr to rgb 
    float left_t = left_c2;
    left_c2 = left_c0;
    left_c0 = left_t;

    float right_t = right_c2;
    right_c2 = right_c0;
    right_c0 = right_t;

    //rgbrgbrgb to rrrgggbbb
    int area = width * height;
    float* pdst_c0 = dst + dy * width + dx;
    float* pdst_c1 = pdst_c0 + area;
    float* pdst_c2 = pdst_c1 + area;
    *pdst_c0 = left_c0;
    *pdst_c1 = left_c1;
    *pdst_c2 = left_c2;

    pdst_c0 = dst + dy * width + dx+3*width*height;
    pdst_c1 = pdst_c0 + area;
    pdst_c2 = pdst_c1 + area;
    *pdst_c0 = right_c0;
    *pdst_c1 = right_c1;
    *pdst_c2 = right_c2;

}

__global__ void Net_reprojectImageTo3D_kernel(uint8_t* left_img,float*disparity,float*pointcloud,float*Q_device,int disparity_rows,int disparity_cols)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid>=disparity_cols*disparity_rows)
    {
        return;
    }
    int col=tid%disparity_cols;
    int row=tid/disparity_cols;

    uint8_t* v = left_img + row * disparity_cols*3 + col * 3;
    float w=Q_device[14]*disparity[row*disparity_cols+col];

    for (size_t i = 0; i < 3; i++)
    {
        pointcloud[(row*disparity_cols+col)*6+i]=(Q_device[i*4]*col+Q_device[i*4+1]*row+Q_device[i*4+3]*1)/w;  
    }
    pointcloud[(row*disparity_cols+col)*6+3]=(float)v[2]; 
    pointcloud[(row*disparity_cols+col)*6+4]=(float)v[1];
    pointcloud[(row*disparity_cols+col)*6+5]=(float)v[0];
}



void Net_preprocess(uint8_t* left,uint8_t* right,int width, int height, float* dst, hipStream_t stream) 
{

    int jobs = width * height;
    int threads = 256;
    int blocks = (jobs +threads-1)/threads;
    Net_preprocess_kernel<<<blocks, threads, 0, stream>>>(left,right,width,height,dst);
}

void Net_reprojectImageTo3D(uint8_t* left_img,float*disparity,float*pointcloud,float*Q_device,int disparity_rows,int disparity_cols)
{
    int jobs=disparity_rows*disparity_cols;
    int threads=256;
    int blocks=(jobs+threads-1)/threads;
    Net_reprojectImageTo3D_kernel<<<blocks,threads>>>(left_img,disparity,pointcloud,Q_device,disparity_rows,disparity_cols);
}
